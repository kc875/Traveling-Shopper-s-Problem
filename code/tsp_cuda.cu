/* CUDA and sequential implemention of the Traveling Shopper Problem*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <algorithm>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_ITEMS 6
#define NUM_NODES 18
#define NUM_NODES_PER_ITEM 3
#define DEBUG false

#define GRID_SIZE 136 //based on 2*SM(68 for 2080 ti)
#define BLOCK_SIZE 1024 //maximum for 2080 ti

using namespace std;





////////////////////////////////////
//        HELPER FUNCTIONS        //
////////////////////////////////////
bool rev(int i, int j) {
    return (i>j);
}


int factorial(int i) {
    if(i == 0) return 0;
    if(i == 1) return 1;
    return i * factorial(i-1);
}


double time_me(timeval start) {
    timeval end;
    gettimeofday(&end, NULL);

    double time_taken = (end.tv_sec - start.tv_sec) * 1e6;
    time_taken = (time_taken + (end.tv_usec - start.tv_usec)) * 1e-6;

    return time_taken;
}


void print_arr(int * arr, int len = NUM_ITEMS) {
    for (int i = 0; i < len; i++) {
        if (arr[i]!=-1) cout<<arr[i]<<' ';
    }
    cout<<endl;
}


bool contains(vector<int> arr, int num) {
    for (int i = 0; i < arr.size(); i++) {
        if (arr[i] == num) return true;
    }
    return false;
}


bool contains_arr(vector<vector<int>> arr, vector<int> l) {
    for (int i = 0; i < arr.size(); i++) {
        if (arr[i].size() != l.size()) {
            break;
        }

        bool match = true;
        for (int j = 0; j < arr[i].size(); j++) {
            if (arr[i][j] != l[j]) {
                match = false;
                break;
            }
        }
        if (match) return true;
    }
    return false;
}



vector<vector<int>> generate_permutations(vector<int> list) {
    vector<vector<int>> output(0);
    if (list.size() == 0) {
        return output;
    }
    if (list.size() == 1 || list[0] == -1) {
        output.push_back(list);
        return output;
    }
    for (int i = 0; i < list.size(); i++) {
        int start = list[i];
        if (start != -1) {
            vector<int> t(0);
            for (int j = 0; j < list.size(); j++) {
                t.push_back(list[j]);
            } 
            t.erase(t.begin()+i);

            vector<vector<int>> temp = generate_permutations(t);
            for (int j = 0; j < temp.size(); j++) {
                t.clear();
                t.push_back(start);
                for (int k = 0; k < temp[j].size(); k++) {
                    t.push_back(temp[j][k]);
                }
                output.push_back(t);
            }
        }
    }
    return output;
}




////////////////////////////////////
//           CUDA STUFF           //
////////////////////////////////////
///////////////////////
// PROBLEM GENERATION//
///////////////////////
__global__ void c_adjacency_matrix(int* out, int size, int seed) {
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    
    if (tidx < tidy) {
        hiprandState_t state;
        hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
            tidx*size+tidy, /* the sequence number is only important with multiple cores */
            0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            &state);
        int val = hiprand(&state)%10+1;
        *(out + tidx*size + tidy) = val;
        *(out + tidy*size + tidx) = val;
    }
    else if (tidx == tidy) {
        *(out + tidx*size + tidy) = 0;
    }
}



__global__ void c_generate_items(int *out, int nodes, int items, int nodes_per_item, int seed) {
    // 2 shops per item
    int tid = threadIdx.x;
    for (int i = 0; i < nodes_per_item; i++) {
        out[nodes_per_item*tid+i] = 0;
    }


    hiprandState_t state;
    hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
        tid, /* the sequence number is only important with multiple cores */
        0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
        &state);
    
    //generating
    for (int i = 0; i < nodes_per_item; i++) {
        int r = (hiprand(&state) % (nodes-1)) + 1;
        bool same = false;
        do {
            same = false;
            for (int j = i-1; j >= 0; j--) {
                if (out[nodes_per_item*tid+j] == r) {
                    same = true;
                    r = (hiprand(&state) % (nodes-1)) + 1;
                    break;
                }
            }
        } while (same);

        out[nodes_per_item*tid+i] = r;
    }

}





///////////////////////
//  PATH GENERATION  //
///////////////////////
__global__ void c_generate_paths(int* item_list, int * out, int nodes, int items, int nodes_per_item) {
    int tid = threadIdx.x;

    int num = tid;
    for (int i = items-1; i >= 0; i--) {
        int j = (items-1) - i;
        int choice = num/(int)(pow(nodes_per_item,i));
        out[tid*items + j] = item_list[nodes_per_item*j + choice];
        num = num%((int)pow(nodes_per_item,i));
    }

}



__device__ void c_permutation(int * path, int * output, int len, int count) {
    for (int i = 0; i<len; i++) {
        output[i] = path[i];
    }
}

__global__ void c_path_permutation(int n, int * paths, int *o_arr, int size, int * shuffle) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) { 
        for (int j = 0; j < size * NUM_ITEMS; j++) {
            o_arr[i*size*NUM_ITEMS+j] = -1;
        }
        //TODO: fill output with permutations of array
        for (int j = 0; j < size; j++) {
            for (int k = 0; k < NUM_ITEMS; k++) {
                o_arr[i*size*NUM_ITEMS+j*NUM_ITEMS+k] = paths[i*NUM_ITEMS+shuffle[j*NUM_ITEMS+k]];
            }
        }
        /*for(int j = 0; j < size; j++){
            for(int k = 0; k < NUM_ITEMS; k++){
                o_arr[i*size*NUM_ITEMS+j*NUM_ITEMS+k] = paths[i*NUM_ITEMS+k];
            }
        } */

        //TODO: check output and remove invalid permutations
        for (int j = 0; j < size; j++) {
            bool valid = true;
            bool neg = true;
            for (int k = NUM_ITEMS-1; k >= 0; k--) {
                if (o_arr[i*size*NUM_ITEMS+j*NUM_ITEMS+k]==-1) {
                    if (!neg) {
                        valid = false;
                    }
                }
                else {
                    neg = false;
                }
            }
            if (!valid) {
                //TODO: erase row (make all -1)
                for (int k = 0; k < NUM_ITEMS; k++) {
                    o_arr[i*size*NUM_ITEMS+j*NUM_ITEMS+k] = -1;
                }
            }
        }
    }
}





///////////////////////
//    PATH LENGTHS   //
///////////////////////
__global__ void c_path_length(int n, int * paths, int *adjacency, int *o_arr, int start_end = 0) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) {
        o_arr[i] = 0;
        if (paths[i*NUM_ITEMS]==-1) {
            o_arr[i] = 10000000;         
        }
        else { 
            o_arr[i] += adjacency[start_end*NUM_NODES + paths[i*NUM_ITEMS+0]];
            int j = 0;
            for (j = 0; j < NUM_ITEMS-1; j++) {
                if (paths[i*NUM_ITEMS+j+1]==-1) {
                    break;          
                }
                o_arr[i] += adjacency[paths[i*NUM_ITEMS+j]*NUM_NODES + paths[i*NUM_ITEMS+j+1]];
            }
            o_arr[i] += adjacency[paths[i*NUM_ITEMS+j]*NUM_NODES + start_end];
        }
    }
}





///////////////////////
//   SHORTEST PATH   //
///////////////////////
__global__ void block_reduce_min(int n, int *i_arr, int *o_arr, int *o_index, bool first = true) {
    int tid = threadIdx.x;
    int sid = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    const int iter = BLOCK_SIZE * gridDim.x;
  
    int min;
    int min_i;
    bool set = false;
    for (int i = sid; i < n; i += iter) {
        if (!set) {
            min = i_arr[i];
            min_i = i;
            set = true;
        }
        if (min > i_arr[i]) {
            min = i_arr[i];
            min_i = i;
        }
    }

    __shared__ int s_arr[BLOCK_SIZE];
    __shared__ int s_index[BLOCK_SIZE]; 
    s_arr[tid] = min;
    s_index[tid] = min_i;
    __syncthreads();
  
    for (int size = BLOCK_SIZE/2; size > 0; size /= 2) {
        if (tid < size) {
            if(s_arr[tid] > s_arr[tid + size]) {
                s_arr[tid] = s_arr[tid + size]; 
                s_index[tid] = s_index[tid + size];
            }
        }
        __syncthreads();
    }
    if (tid == 0) {
        o_arr[blockIdx.x] = s_arr[0];
        if(first){
            o_index[blockIdx.x] = s_index[0];
        }
        else{
            o_index[blockIdx.x] = o_index[s_index[0]];
        }
    }
}



__global__ void c_find_min(int size, int * c_lengths, int * c_out, int * c_index) {
    int thread = threadIdx.x;
    int num_threads = blockDim.x;

    int min = 100000;
    int id = -1;

    for (int i = thread; i < size; i += num_threads) {
        if (c_lengths[i] < min && c_lengths[i] > 0) {
            min = c_lengths[i];
            id = i;
        }
    }

    c_out[thread] = min;
    c_index[thread] = id;
    __syncthreads();

    for (int part_size = num_threads/2; part_size > 0; part_size /= 2) {
        if (thread < part_size) {
            if (c_out[part_size + thread] < c_out[thread] && c_out[part_size + thread] > 0 && c_index[part_size + thread] > 0) {
                c_out[thread] = c_out[part_size + thread];
                c_index[thread] = c_index[part_size + thread];
            }
            __syncthreads();
        }
    }

}





///////////////////////
//   CUDA FUNCTION   //
///////////////////////  
void ts_cuda() {
    struct timeval start;

    // problem generation
    gettimeofday(&start, NULL);

    cout<<"---Starting cuda calculation---"<<endl;

    cout<<"Generating problem"<<endl;
    int * c_adjacency;
    int * c_items;
    hipMalloc(&c_adjacency, NUM_NODES*NUM_NODES*sizeof(int));
    hipMalloc(&c_items, NUM_NODES_PER_ITEM*NUM_ITEMS*sizeof(int));

    c_adjacency_matrix<<<1, dim3(NUM_NODES, NUM_NODES)>>>(c_adjacency, NUM_NODES, (int)time(0));

    c_generate_items<<<1, NUM_ITEMS>>>(c_items, NUM_NODES, NUM_ITEMS, NUM_NODES_PER_ITEM, (int)time(0));

    cout<<"Calculating paths"<<endl;
    // partial paths
    int num_paths = (int)pow(NUM_NODES_PER_ITEM, NUM_ITEMS);
    int * c_part_paths;
    hipMalloc(&c_part_paths, num_paths*NUM_ITEMS*sizeof(int));
    c_generate_paths<<<1,num_paths>>>(c_items, c_part_paths, NUM_NODES, NUM_ITEMS, NUM_NODES_PER_ITEM);
    if (DEBUG) {
        int * tmp_out3 = (int*) malloc(num_paths*NUM_ITEMS*sizeof(int));
        hipDeviceSynchronize();
        hipMemcpy(tmp_out3, c_part_paths, num_paths*NUM_ITEMS*sizeof(int), hipMemcpyDeviceToHost);
        for (int i = 0; i < num_paths; i++) {
            for (int j = 0; j < NUM_ITEMS; j++) {
                cout<<tmp_out3[i*NUM_ITEMS + j]<<" ";
            }
            cout<<endl;
        }
        cout<<endl;
        free(tmp_out3);
    }

    
    // paths
    int size = num_paths;
    int * c_paths;
    int f = factorial(NUM_ITEMS);
    int * c_shuffle;

    vector<int> t;
    for (int i = 0; i < NUM_ITEMS; i++) {
        t.push_back(i);
    }
    vector<vector<int>> temp = generate_permutations(t);
    t.clear();
    for (int i = 0; i < temp.size(); i++) {
        for (int j = 0; j < temp[i].size(); j++) {
            t.push_back(temp[i][j]);
        }
        //print_arr(&temp[i][0]);
    }
    
    hipMalloc(&c_shuffle, f*NUM_ITEMS*sizeof(int));
    hipMemcpy(c_shuffle, &t[0], f*NUM_ITEMS*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&c_paths, num_paths*NUM_ITEMS*f*sizeof(int));

    c_path_permutation<<<(size + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(size, c_part_paths, c_paths, f, c_shuffle);
    if (DEBUG) {
        int * tmp_out2 = (int*) malloc(num_paths*NUM_ITEMS*f*sizeof(int));
        hipDeviceSynchronize();
        hipMemcpy(tmp_out2, c_paths, num_paths*NUM_ITEMS*f*sizeof(int), hipMemcpyDeviceToHost);
        for (int i = 0; i < num_paths*f; i++) {
            for (int j = 0; j < NUM_ITEMS; j++) {
                cout<<tmp_out2[i*NUM_ITEMS + j]<<" ";
            }
            cout<<endl;
        }
        cout<<endl;
        free(tmp_out2);
    }



    cout<<"Calculate path lengths"<<endl;
    size = num_paths*f;
    int * c_lengths;
    hipMalloc(&c_lengths, size*sizeof(int));

    c_path_length<<<(size + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(size, c_paths, c_adjacency, c_lengths);
    if (DEBUG) {
        int * tmp_out1 = (int*) malloc(size*sizeof(int));
        hipDeviceSynchronize();
        hipMemcpy(tmp_out1, c_lengths, size*sizeof(int), hipMemcpyDeviceToHost);
        for (int i = 0; i < size; i++) {
            cout<<tmp_out1[i]<<" ";
        }
        cout<<endl;
        free(tmp_out1);
        cout<<size<<" paths\n"<<endl;
    }

    hipDeviceSynchronize();

    cout<<"Finding shortest path"<<endl;

    int *c_out;
    int *c_index;
    hipMalloc(&c_out, sizeof(int) * BLOCK_SIZE);  
    hipMalloc(&c_index, sizeof(int) * BLOCK_SIZE);


    c_find_min<<<1,BLOCK_SIZE>>>(size, c_lengths, c_out, c_index);

    hipDeviceSynchronize();

    int index;
    int min;
    int *path = (int*)malloc(num_paths*NUM_ITEMS*f*sizeof(int));
    
    hipMemcpy(&index, c_index, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&min, c_out, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(path, c_paths, num_paths*NUM_ITEMS*f*sizeof(int), hipMemcpyDeviceToHost);
    
    double etime = time_me(start);
    cout<<"Shortest Path Length: "<<min<<"\nIndex: "<<index<<endl;
    print_arr(&path[index*NUM_ITEMS]);

    cout<<"Time Taken (cuda): "<<fixed<<setprecision(6)<<etime<<"\n"<<endl;

    hipFree(c_adjacency);
    hipFree(c_items);
    hipFree(c_part_paths);
    hipFree(c_paths);
    hipFree(c_lengths);
    hipFree(c_out);
    hipFree(c_index);
    free(path);
}



int main(void) {
    ts_cuda();
}
